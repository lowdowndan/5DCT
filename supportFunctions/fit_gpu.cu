#include "hip/hip_runtime.h"
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <math.h>
#include <iostream>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"


// Mandatory global methods for C++ support
//extern "C" void __cxa_pure_virtual()
//{
//    // Do nothing or print an error message.
//}
////void *__dso_handle = 0;
//extern "C" int __cxa_atexit(void (*destructor) (void *), void *arg, void *dso)
//{
//    //arg;
//    //dso;
//    return 0;
//}
//extern "C" void __cxa_finalize(void *f)
//{ //f;
//}



// cuda error checking
#define cudaCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess) 
   {
      mexErrMsgIdAndTxt("MATLAB:hipError_t","Error: %s \n In file %s at line %d.\n", hipGetErrorString(code), file, line);
   }
}

// cuBLAS error checking
#define cublasCheck(ans) { gpuAssert_cublas((ans), __FILE__, __LINE__); }
inline void gpuAssert_cublas(hipblasStatus_t code, const char *file, int line)
{
   if (code != HIPBLAS_STATUS_SUCCESS) 
   {
      mexErrMsgIdAndTxt("MATLAB:cublasError","cuBLAS error: %s \n In file %s at line %d.\n", _cudaGetErrorEnum(code), file, line);
   }
}


void cleanup(){

	hipDeviceReset();
}

// Use the z component of DVFs for linear interpolation of surrogate (track each voxel)
__global__ void deform_surrogate_kernel(hipTextureObject_t vTex, hipTextureObject_t fTex, float* hd_registration, float* hd_X, float* hd_X2, int nSlices, int nScans, int nVoxels, int nx, int ny, int nz, int startSlice){

	// Calculate indices
	
	// Linear index -- what voxel am i? 
	int ind = (blockIdx.x * blockDim.x) + threadIdx.x;

	// Index into X matrix (1st column for this voxel)
	int xInd = ind * nScans * 3;

	// Index into z component registration matrix 
	int rInd = (ind * nScans * 3) + (nScans * 2);

	// Bounds check
	if (ind > (nVoxels - 1)){
		return;
	}
	
	// Get the slice number of this thread.  Integer division here is intentional
	int z = (startSlice - 1) + (ind / (nx * ny));
	float z2;

	// Interpolate to get voltage and flow from each scan by applying the z
	// component of the DVF for this scan

	float v;
	float f;
	float dz;

	for(int i = 0; i < nScans; i++){

	// Get deformed z coordinate
	dz = hd_registration[rInd + i];
	z2 = (float)z + dz; 

	// CUDA uses row major ordering, so indexing is transosed from matlab v matrix
	v = tex2D<float>(vTex, (z2 + 0.5f), ((float) i + 0.5f));
	f = tex2D<float>(fTex, (z2 + 0.5f), ((float) i + 0.5f));

	// Write output
	hd_X[xInd + i ] = 1.0f;
	hd_X[xInd + i + (1 * nScans)] = v;
	hd_X[xInd + i + (2 * nScans)] = f;

	hd_X2[xInd + i ] = 1.0f;
	hd_X2[xInd + i + (1 * nScans)] = v;
	hd_X2[xInd + i + (2 * nScans)] = f;

	}
}


// Set thread block size
#define BLOCKWIDTH 512
//#define BLOCKHEIGHT 16 

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[]){

// input
#define m_vSlices prhs[0]
#define m_fSlices prhs[1]
#define m_dim prhs[2]
#define m_startSlice prhs[3]
#define m_registration prhs[4]

// output
//#define m_debug plhs[0]
#define m_parameters plhs[0]
#define m_model plhs[1]
//#define m_residual plhs[2]

const static int N_PARAMETERS = 3;
const static int N_DIMS = 3;

// rows and columns

int nScans = (int) mxGetM(m_registration);
int nVoxels = (int) mxGetN(m_registration) / N_DIMS;
int nSlices = (int) mxGetM(m_vSlices);

// slice dimensions
double* hh_dim = (double*) mxGetData(m_dim);
int nx = (int) hh_dim[0];
int ny = (int) hh_dim[1];
int nz = (int) hh_dim[2];

// Get the starting slice of this chunk
double* hh_startSlice = (double*) mxGetData(m_startSlice);
int startSlice = (int) *hh_startSlice;

// cuBLAS initialization
hipblasStatus_t cublasStat;

hipblasHandle_t handle;
cublasStat = hipblasCreate(&handle);
cublasCheck(cublasStat);


// Memory allocation sizes
size_t nBytesRegistration = nScans * nVoxels * N_DIMS * sizeof(float);
size_t nBytesX = nBytesRegistration;
size_t nBytesSurrogate = nScans * nSlices * sizeof(float);

// Host memory for info matrix
int * info = (int *)malloc(nVoxels * sizeof(int));

// Get pointers to host data
float* hh_registration = (float*) mxGetData(m_registration);
float* hh_vSlices = (float*) mxGetData(m_vSlices);
float* hh_fSlices = (float*) mxGetData(m_fSlices);


// Allocate and copy
hipArray* hda_vSlices;
hipArray* hda_fSlices;

float* hd_registration;
float* hd_X;
float* hd_X2;

cudaCheck(hipMalloc((void**)&hd_registration, nBytesRegistration));
cudaCheck(hipMalloc((void**)&hd_X, nBytesX));
cudaCheck(hipMalloc((void**)&hd_X2, nBytesX));

cudaCheck(hipMemcpy(hd_registration,hh_registration, nBytesRegistration, hipMemcpyHostToDevice));

// Copy v,f slice measurements to textures

hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

cudaCheck(hipMallocArray(&hda_vSlices, &channelDesc,nSlices, nScans));
cudaCheck(hipMallocArray(&hda_fSlices, &channelDesc,nSlices, nScans));

cudaCheck(hipMemcpyToArray(hda_vSlices,0,0,hh_vSlices,nBytesSurrogate,hipMemcpyHostToDevice)); 
cudaCheck(hipMemcpyToArray(hda_fSlices,0,0,hh_fSlices,nBytesSurrogate,hipMemcpyHostToDevice));

// Create texture objects

// v
hipResourceDesc resDesc;
memset(&resDesc, 0, sizeof(resDesc));
resDesc.resType = hipResourceTypeArray;

hipTextureDesc texDesc;
memset(&texDesc, 0, sizeof(texDesc));
texDesc.addressMode[0] = hipAddressModeClamp;
texDesc.addressMode[1] = hipAddressModeClamp;
texDesc.filterMode = hipFilterModeLinear;
texDesc.readMode = hipReadModeElementType;
texDesc.normalizedCoords = 0;

hipTextureObject_t vTex = 0;
resDesc.res.array.array = hda_vSlices;
hipCreateTextureObject(&vTex, &resDesc, &texDesc, NULL);

// f
resDesc.res.array.array = hda_fSlices;
hipTextureObject_t fTex = 0;
hipCreateTextureObject(&fTex, &resDesc, &texDesc, NULL);


const dim3 blockSize(BLOCKWIDTH);
const dim3 gridSize((nVoxels/BLOCKWIDTH) + 1);

//float* hd_buffer;
//int nBytesBuffer = nx * ny * sizeof(float);
//cudaCheck(hipMalloc((void**)&hd_buffer, nBytesBuffer));

// Deform bellows by z vector of dvf, calculate appropriate v and f for all slices
deform_surrogate_kernel<<<gridSize,blockSize>>>(vTex, fTex, hd_registration, hd_X, hd_X2, nSlices, nScans, nVoxels, nx, ny, nz, startSlice);
cudaCheck(hipDeviceSynchronize());

//return;
//m_debug = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS,mxREAL);
//mxSetM(m_debug, nx);
//mxSetN(m_debug, ny);
//mxSetData(m_debug, mxMalloc(nBytesBuffer));
//float * hh_debug = (float*) mxGetData(m_debug);
//cudaCheck(hipMemcpy(hh_debug,hd_buffer,(nBytesBuffer),hipMemcpyDeviceToHost));
//return;

// Host pointer arrays to device data
float ** hda_X = (float **)malloc(nVoxels * sizeof(float*));
float ** hda_X2 = (float **)malloc(nVoxels * sizeof(float*));
float ** hda_registration = (float **)malloc(nVoxels * sizeof(float*));

// Set arrays of pointers to submatrices
for (int i = 0; i < nVoxels; i++){
	hda_X[i] = hd_X + (i * nScans * N_DIMS);
	hda_X2[i] = hd_X2 + (i * nScans * N_DIMS);
	hda_registration[i] = hd_registration + (i * nScans * N_DIMS);
}

// Device pointers to device data
float ** dd_X;
float ** dd_X2;
float ** dd_registration;

cudaCheck(hipMalloc((void**)&dd_X, nVoxels * sizeof(float*)));
cudaCheck(hipMalloc((void**)&dd_X2, nVoxels * sizeof(float*)));
cudaCheck(hipMalloc((void**)&dd_registration, nVoxels * sizeof(float*)));

// Copy array of pointers to device
cudaCheck(hipMemcpy(dd_X, hda_X, nVoxels * sizeof(float*), hipMemcpyHostToDevice));
cudaCheck(hipMemcpy(dd_X2, hda_X2, nVoxels * sizeof(float*), hipMemcpyHostToDevice));
cudaCheck(hipMemcpy(dd_registration, hda_registration, nVoxels * sizeof(float*), hipMemcpyHostToDevice));

// Solve
cublasStat = hipblasSgelsBatched(handle, HIPBLAS_OP_N, (int) nScans, (int) N_PARAMETERS, (int) N_DIMS, dd_X, nScans, dd_registration,nScans,info,NULL,nVoxels);
cublasCheck(cublasStat);


// Allocate host memory for parameters
m_parameters = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS,mxREAL);
mxSetM(m_parameters, N_PARAMETERS);
mxSetN(m_parameters, (N_DIMS * nVoxels));
mxSetData(m_parameters, mxMalloc(sizeof(float) * N_PARAMETERS * N_DIMS * nVoxels));
float * h_parameters = (float*) mxGetData(m_parameters);

// Copy parameters to host
cublasStat = hipblasGetMatrix(N_PARAMETERS, (N_DIMS * nVoxels), sizeof(float), hd_registration, nScans, h_parameters, N_PARAMETERS);
cublasCheck(cublasStat);


// Multiply to get fits

// Set constants for matrix Sgemm
float * const alpha = (float *)malloc(sizeof(float));
*alpha = 1.0;

float * const beta = (float *)malloc(sizeof(float));
*beta = 0.0;
	
// Allocate device memory for model fit
float * hd_model;
unsigned int nBytesModel = sizeof(float) * nScans * N_DIMS * nVoxels;
cudaCheck(hipMalloc((void**) &hd_model, nBytesModel));


// Array of pointers to fit submatrices
float ** hda_model = (float **)malloc(nVoxels * sizeof(float*));
float **  dd_model;

for(int i = 0; i < nVoxels; i++){
	hda_model[i] = hd_model + (i * nScans * N_DIMS);
}


// Copy array of fit pointers to device
cudaCheck(hipMalloc((void**)&dd_model, nVoxels * sizeof(float*)));
cudaCheck(hipMemcpy(dd_model,hda_model,nVoxels * sizeof(float*), hipMemcpyHostToDevice));

// Calcluate model fit
cublasStat = hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_T,N_DIMS,nScans,N_DIMS,alpha,(const float **) dd_registration,nScans,(const float **) dd_X2,nScans,beta,dd_model, N_PARAMETERS, nVoxels);
cublasCheck(cublasStat);

// Allocate host memory for fit
m_model = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS,mxREAL);
mxSetM(m_model, N_DIMS);
mxSetN(m_model, (nScans * nVoxels));
mxSetData(m_model, mxMalloc(nBytesModel));
float * h_model = (float*) mxGetData(m_model);

// Copy fit to host
cudaCheck(hipMemcpy(h_model,hd_model,nBytesModel,hipMemcpyDeviceToHost));
cublasStat = hipblasGetMatrix(N_DIMS, (nScans * nVoxels), sizeof(float), hd_model, N_DIMS, h_model, N_DIMS);
cublasCheck(cublasStat);


// Okay, we're done here.  Free memory, close cublas and exit
free(hda_X);
free(hda_X2);
free(hda_registration);
free(hda_model);
free(info);
free(alpha);
free(beta);

hipFreeArray(hda_vSlices);
hipFreeArray(hda_fSlices);

hipFree(hd_X);
hipFree(hd_X2);
hipFree(hd_registration);
hipFree(hd_model);
hipFree(dd_X);
hipFree(dd_X2);
hipFree(dd_registration);
hipFree(dd_model);

hipDestroyTextureObject(vTex);
hipDestroyTextureObject(fTex);

hipblasDestroy(handle);
//free(resDesc);
//free(texDesc);
//free(channelDesc);

//Reset device for profiling
hipDeviceReset();

mexAtExit(cleanup);
//return;
}

